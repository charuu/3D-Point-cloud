
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <time.h>
#include <algorithm>
#include <math.h>


#define SQ(x) ((x)*(x))
extern "C" int* ball_tree_search_wrapper(double* points_c,int* T);
__host__ void eballsearch_next(
       int           *m,  /*  O  |   1   | next index within radius e */
       int           *S,  /*  W  | 2logN | stack                      */
       int           *q,  /*  W  |   1   | top index of stack 'S'     */
       const double  *y,  /*  I  |   D   | the point of interest      */
       double         e,  /*  I  | const.| ball radius                */
       const double  *X,  /*  I  |   DN  | points                     */
       const int     *T,  /*  I  |  3N+1 | kdtree                     */
       int            D,  /*  I  | const.| dimension                  */
       int            N   /*  I  | const.| #points                    */
     );
__host__ static double dist(const double *x, const double *y, double D);
__host__ void kdtree(
       int           *T,  /* O | 3N+1 | depth(N),left(N),right(N),root(1)    */
       int           *a,  /* W |  6N  | index(N),size(N),buffer(3N),stack(N) */
       double        *v,  /* W |  2N  | buffer1(N), buffer2(N)               */
       const double  *X,  /* I |  DN  | points                               */
       int            D,  /* I |      | dimension                            */
       int            N   /* I |      | #points                              */
    );
__host__ int *kdtree_build(const double *X, int D, int N);


extern "C" int* ball_tree_search_wrapper(double* points_c,int* T) {
  //  int *a,*u,*S;
    double* A=points_c;
    double* B=points_c;
    int D=3;
    int N=50000;
   // int *wi;
    //srand(16);
    //int * T ;//=(int *)calloc(3*50000+1,sizeof(int));
   // wi=(int*)calloc(6*50000,  sizeof(int));
    T = kdtree_build(A,3,50000);
    
   // for(int i=0;i<10;i++){
      //  printf("%d, %d, %d\n",T[0],T[1],T[2]);
      //   T = T+3;
   // }
   // int mtd=10;int j;int si=0;
  //  wi+=6*50000; a=wi+si;si+=50000; S=wi+si;si+=mtd*50000;
  //  u=wi+si;si+=50000;
  
 // for(int i=0;i<50000;i++){
  //  a[i]=u[i]=0;
  //  do{ 
      
     // eballsearch_next(a+i,S+mtd*i,u+i,A+D*i,0.001,B,T,D,50000); 
   //   j=a[i];
   //   printf("query:%lf,found:%lf\n",A[D*i],B[D*j]);
     
   // } while(u[i]);
 // } ;
//free(A);free(B);
//free(T);
return T;
}
__host__ void eballsearch_next(
       int           *m,  /*  O  |   1   | next index within radius e */
       int           *S,  /*  W  | 2logN | stack                      */
       int           *q,  /*  W  |   1   | top index of stack 'S'     */
       const double  *y,  /*  I  |   D   | the point of interest      */
       double         e,  /*  I  | const.| ball radius                */
       const double  *X,  /*  I  |   DN  | points                     */
       const int     *T,  /*  I  |  3N+1 | kdtree                     */
       int            D,  /*  I  | const.| dimension                  */
       int            N   /*  I  | const.| #points                    */
     ){

  int d,p,nl,nr,n=T[3*N],state=1; double u,v;

  if(*q==0) S[(*q)++]=n;
  while((*q)&&state){ n=S[--(*q)];
     nl=T[n+N*1]; p=T[n];
     nr=T[n+N*2]; d=p%D;

     if(dist(y,X+D*n,D)<=e){*m=n;state=0;}

     v=y[d]-X[d+D*n]; u=fabs(v);
     if   (v>0){if(nr>=0)S[(*q)++]=nr; if(nl>=0&&u<=e)S[(*q)++]=nl;}
     else      {if(nl>=0)S[(*q)++]=nl; if(nr>=0&&u<=e)S[(*q)++]=nr;}

  }  if( state) *m=-1;
}
__host__ static double dist(const double *x, const double *y, double D){
  int d; double val=0;
  for(d=0;d<D;d++) val+=SQ(x[d]-y[d]);
  return sqrt(val);
}
__host__ static void swap(double *a, double *b){double tmp; tmp=*a; *a=*b; *b=tmp;}

__host__ double median(double *a, double *w, const int N){
  const int c = N/2;
  int     i,j,k,l,u,e,ofs=0,size=N;
  double  *tmp, p;/*pivot*/

  while(1){i=j=k=0;p=a[0];e=1;
    for(i=1;i<size;i++){
      if      (a[i]< p)  a[j++]=a[i];
      else if (a[i]> p)  w[k++]=a[i];
      else   /*a[i]==p*/ e++;
    } l=ofs+j;u=l+e-1;

    if      (c<l) {size=j;}
    else if (c>u) {tmp=a;a=w;w=tmp;ofs=u+1;size=k;}
    else break;
  }

  return p;
}

__host__ static inline void divide(
   int          *  b,    /*  I/O | 5N | array to be divided         */
   double       *  v,    /*   W  | 2N | working memory              */
   int             K,    /*   I  |    | size of array to be divided */
   const double *  X,    /*   I  | DN | points                      */
   int             D,    /*   I  |    | dimension                   */
   int             N,    /*   I  |    | #points                     */
   int             p     /*   I  |    | current depth               */
  ){
  int k,i=0,j=0,e=0,c=K/2,d=p%D; double med,val; double *w=v+N;
  int *sz=b+N,*bl=b+2*N,*bc=b+3*N,*br=b+4*N;

  *sz=K; /* store original array size */
  for(k=0;k<K;k++){v[k]=X[d+D*b[k]];} swap(v,v+K/2);

  if (K==1) bl[0]=b[c];
  else { med=median(v,w,K);
    for(k=0;k<K;k++){ val=X[d+D*b[k]];
      if      (val<med) bl[i++]=b[k];
      else if (val>med) br[j++]=b[k];
      else              bc[e++]=b[k];
    }
    for(k=0;k<i;k++) b[k    ]=bl[k];
    for(k=0;k<e;k++) b[k+i  ]=bc[k];
    for(k=0;k<j;k++) b[k+i+e]=br[k];
  }
}
__host__ int *kdtree_build(const double *X, int D, int N){
   int *T,*wi; double *wd;

  T =(int *)calloc(3*N+1,sizeof(int));
  wi=(int *)calloc(6*N,  sizeof(int));
  wd=(double *)calloc(2*N,  sizeof(double));
  kdtree(T,wi,wd,X,D,N);
  free(wi); free(wd);

  return T;
}
__host__ void kdtree(
       int           *T,  /* O | 3N+1 | depth(N),left(N),right(N),root(1)    */
       int           *a,  /* W |  6N  | index(N),size(N),buffer(3N),stack(N) */
       double        *v,  /* W |  2N  | buffer1(N), buffer2(N)               */
       const double  *X,  /* I |  DN  | points                               */
       int            D,  /* I |      | dimension                            */
       int            N   /* I |      | #points                              */
    ){
  int *bl=NULL,*br=NULL,*b=a,*S=a+5*N; int n,nl,nr,q=0,c,cl,cr,s,sl,sr; int p=0;

  /* init */
  for(n=0;n<3*N;n++) T[n]=-1;
  for(n=0;n<  N;n++) b[n]= n;
  /* basis */
  divide(b,v,N,X,D,N,p); S[q++]=0;c=N/2;T[b[c]]=p; T[3*N]=b[c]; /*root*/
  
  /* step */
  while(q){ b=a+S[--q];s=*(b+N); /*pop*/
    /* compute locations and sizes of divided arrays */
    c=s/2; n=b[c]; bl=b; br=bl+c+1; sl=c; sr=c-(s%2?0:1); p=T[n];
    /* division of coordinate space */
    if(sl){divide(bl,v,sl,X,D,N,p+1);S[q++]=bl-a;cl=sl/2;nl=bl[cl];T[nl]=p+1;T[n+N*1]=nl;}
    if(sr){divide(br,v,sr,X,D,N,p+1);S[q++]=br-a;cr=sr/2;nr=br[cr];T[nr]=p+1;T[n+N*2]=nr;}
  }
}
