

#include <hip/hip_runtime.h>
#include <stdio.h>


void __global__ print()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("I am in GPU");
}

extern "C" void f()
{
    print<<<1, 10>>>();
    hipDeviceSynchronize();
     return;
}
