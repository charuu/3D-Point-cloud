#include <stdio.h>

#include <stdlib.h>

#include <string.h>

#include <hip/hip_runtime.h>


extern "C" void kernel_wrapper(int *a);

__global__ void kernel(int *a)

{

    int tx = threadIdx.x;

    

    switch( tx )

    {

	case 0:

     a[tx] = a[tx] + 2;

     break;

	case 1:

     a[tx] = a[tx] + 3;

     break;

    }

}

void kernel_wrapper(int *a)

{
 
    int *d_a;
    int size = 256;
   // dim3 threads( 2, 1 );

   // dim3 blocks( 1, 1 );

    hipMalloc( (void **)&d_a, sizeof(int) * (size_t)size );

    hipMemcpy( d_a, a, sizeof(int) * size, hipMemcpyDeviceToHost );


    printf( "Finish kernel wrapper\n" );

    hipFree(d_a);

}