#ifndef __B_H_

#define __B_H_

#include <stdio.h>

#include <stdlib.h>

#include <string.h>

#include <hip/hip_runtime.h>



#endif

extern "C" void kernel_wrapper(int *a);

__global__ void kernel(int *a)

{

    int tx = threadIdx.x;

    

    switch( tx )

    {

	case 0:

     a[tx] = a[tx] + 2;

     break;

	case 1:

     a[tx] = a[tx] + 3;

     break;

    }

}

void kernel_wrapper(int *a)

{
 
    int *d_a;

    dim3 threads( 2, 1 );

    dim3 blocks( 1, 1 );

   hipMalloc( (void **)&d_a, sizeof(int) * 2 );

   hipMemcpy( d_a, a, sizeof(int) * 2, hipMemcpyHostToDevice );

   kernel<<< blocks, threads >>>( d_a );

   hipMemcpy( a, d_a, sizeof(int) * 2, hipMemcpyDeviceToHost );

   printf( "Finish kernel wrapper\n" );

    hipFree(d_a);

}